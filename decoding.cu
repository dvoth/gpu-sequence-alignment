
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include <inttypes.h>
#include <bitset>

//takes a size K, and a char array
__host__ __device__ void decodeData(long long val, int KSize, char *result) {
	//k defines the number of bits revelant to the bit mask
	// k must be strictly less than 32 for now, since a long long is 64 bits 
	/*
		A = 00;
		C = 01;
		G = 10;
		T = 11;

	*/
	//assume the largest K which is 32 and creates an array for it
	//will have to chang when we can view larger K values
	//since this can't be dynamic, that is why it is 32
	for (int i = 0; i < KSize; i++) {
		long long unsigned one = 1;
		long long unsigned temp1 = one & val;
		val = val >> 1;
		long long unsigned temp2 = one & val;
		val = val >> 1;
		if (temp2 == 0 && temp1 == 0)
			result[KSize - i - 1] = 'A';
		else if (temp2 == 0 && temp1 == 1)
			result[KSize - i - 1] = 'C';
		else if (temp2 == 1 && temp1 == 0)
			result[KSize - i - 1] = 'G';
		else
			result[KSize - i - 1] = 'T';
	}
}