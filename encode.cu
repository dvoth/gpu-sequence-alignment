#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <inttypes.h>
#include <bitset>
#include "encodedData.cuh"
#include "Decode.cu"


const int maxKSize = 105;

__host__ __device__ void baseToLong(char *val, int KSize, struct encodedData * result){
   /*
		A = 000;
		C = 001;
		G = 010;
		T = 011;
		N = 100;
	*/
	long long unsigned oneflip = 1;
	//the struct holds 320 bits all together over 5 different longs
	//while only a single long long can hold 64 bits and the encoding comes out to 3 bits per base to encode all 5
	//thus it only holds 21 bases per long long when rounded down (21.3333_ when not rounded)
	//this means the max k can be 105 since there will be a single bit not usede in each long long

	long long unsigned *encodingTemp = NULL;
	result->getNextEmptyLong(&encodingTemp);

	for (int i = 0; i < KSize; i++) {
		//if true, them change to the other version of the struct
		if (i % 21 == 0 && i > 0)
			result->getNextEmptyLong(&encodingTemp);

		int aciiVal = int(val[i]);
		//A val - 000
		if (aciiVal == 65 || aciiVal == 97) {
			//shift left 3 times
			(*encodingTemp) = (*encodingTemp) << 1;
			(*encodingTemp) = (*encodingTemp) << 1;
			(*encodingTemp) = (*encodingTemp) << 1;
		}
		//C val - 100
		else if (aciiVal == 67 || aciiVal == 99) {
			//shift left 3 times
			(*encodingTemp) = (*encodingTemp) << 1;
			(*encodingTemp) = (*encodingTemp) << 1;
			(*encodingTemp) = (*encodingTemp) << 1;
			//flip last bit
			(*encodingTemp) = (*encodingTemp) | oneflip;

		}
		//G val - 010
		else if (aciiVal == 71 || aciiVal == 103) {
			//shift left once
			(*encodingTemp) = (*encodingTemp) << 1;
			(*encodingTemp) = (*encodingTemp) << 1;
			//flip first 1
			(*encodingTemp) = (*encodingTemp) | oneflip;
			//shit left again twice
			//(*encodingTemp) = (*encodingTemp) << 1;
			(*encodingTemp) = (*encodingTemp) << 1;
		}
		//T val - 110
		else if (aciiVal == 84 || aciiVal == 116) {
			//shift left twice
			(*encodingTemp) = (*encodingTemp) << 1;
			(*encodingTemp) = (*encodingTemp) << 1;
			//flip first 1
			(*encodingTemp) = (*encodingTemp) | oneflip;
			//shift left once
			(*encodingTemp) = (*encodingTemp) << 1;
			//flip first 1
			(*encodingTemp) = (*encodingTemp) | oneflip;
		}
		//default N value - 001
		else {
			//shift once
			(*encodingTemp) = (*encodingTemp) << 1;
			//flip the bit 
			(*encodingTemp) = (*encodingTemp) | oneflip;
			//shift twice
			(*encodingTemp) = (*encodingTemp) << 1;
			(*encodingTemp) = (*encodingTemp) << 1;
		}
	}


}

__host__ __device__ void encodeData(char *val, int KSize){ 


	/*
		A = 000;
		C = 001;
		G = 010;
		T = 011;
		N = 100;
	
	*/
    
    //create an L-Mer which is of size K (defined by user or default)
	struct encodedData lMer;
	int LSize = KSize;              		//reduce the size sine L = K - 1
	baseToLong(val, LSize, (&lMer));       //transform the bases into the long long representation


	//create a K-Mer which is size K-1
	struct encodedData kMer;                //create the var that will be the K-Mer
	int KSize1 = LSize - 1;                 //reduce size
	baseToLong(val, KSize1, (&kMer));       //transform the bases into the long long representation

    //next step is to put the encoded K-mer into the hash table
}


