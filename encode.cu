
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include <inttypes.h>
#include <bitset>
__host__ __device__ void encodData(char *val, int KSize){
    //k defines the number of bits revelant to the bit mask
    // k must be strictly less than 32 for now, since a long long is 64 bits 
    /*
        A = 00;
        C = 01;
        G = 10;
        T = 11;
    
    */
    long long unsigned oneflip = 1;
    long long unsigned result = 0;
    if (KSize <= 32)
    {
        for (int i = 0; i < KSize; i++) {
            int aciiVal = int(val[i]);
            //A val
            if (aciiVal == 65 || aciiVal == 97) {
                //shift left twice
                result = result << 1;
                result = result << 1;
            }
            //C val
            else if (aciiVal == 67 || aciiVal == 99) {
                //shift left twice
                result = result << 1;
                result = result << 1;
                //flip last bit
                result = result | oneflip;

            }
            //G val
            else if (aciiVal == 71 || aciiVal == 103) {
                //shift left once
                result = result << 1;
                //flip first 1
                result = result | 1;
                //shit left again
                result = result << 1;
            }
            //T val
            else if (aciiVal == 84 || aciiVal == 116) {
                //shift left once
                result = result << 1;
                //flip first 1
                result = result | 1;
                //shift left once
                result = result << 1;
                //flip first 1
                result = result | 1;
            }
            //default bad val
            else{
                printf("Bad Input Flag Triggered \n");
                break;
            }
        }
    }
    //next step is to put the encoded K-mer into the hash table
}