#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <inttypes.h>
#include <bitset>

#include "encodedData.cuh"
#include "Decode.cu"
#include "mer_list.cuh"

const int maxKSize = 105;

__host__ __device__ void baseToLong(char *val, int KSize, struct encodedData * result){
   /*
		A = 000;
		C = 001;
		G = 010;
		T = 011;
		N = 100;
	*/
	long long unsigned oneflip = 1;
	//the struct holds 320 bits all together over 5 different longs
	//while only a single long long can hold 64 bits and the encoding comes out to 3 bits per base to encode all 5
	//thus it only holds 21 bases per long long when rounded down (21.3333_ when not rounded)
	//this means the max k can be 160 
	//this will guarntee that K <= 105 which
	if (KSize > 105)
		return;

	long long unsigned *encodingTemp = NULL;
	result->getNextEmptyLong(&encodingTemp);

	for (int i = 0; i < KSize; i++) {
		//if true, them change to the other version of the struct
		if (i % 21 == 0 && i > 0)
			result->getNextEmptyLong(&encodingTemp);

		int aciiVal = int(val[i]);
		//A val - 000
		if (aciiVal == 65 || aciiVal == 97) {
			//shift left 3 times
			(*encodingTemp) = (*encodingTemp) << 1;
			(*encodingTemp) = (*encodingTemp) << 1;
			(*encodingTemp) = (*encodingTemp) << 1;
		}
		//C val - 100
		else if (aciiVal == 67 || aciiVal == 99) {
			//shift left 3 times
			(*encodingTemp) = (*encodingTemp) << 1;
			(*encodingTemp) = (*encodingTemp) << 1;
			(*encodingTemp) = (*encodingTemp) << 1;
			//flip last bit
			(*encodingTemp) = (*encodingTemp) | oneflip;

		}
		//G val - 010
		else if (aciiVal == 71 || aciiVal == 103) {
			//shift left once
			(*encodingTemp) = (*encodingTemp) << 1;
			(*encodingTemp) = (*encodingTemp) << 1;
			//flip first 1
			(*encodingTemp) = (*encodingTemp) | oneflip;
			//shit left again twice
			//(*encodingTemp) = (*encodingTemp) << 1;
			(*encodingTemp) = (*encodingTemp) << 1;
		}
		//T val - 110
		else if (aciiVal == 84 || aciiVal == 116) {
			//shift left twice
			(*encodingTemp) = (*encodingTemp) << 1;
			(*encodingTemp) = (*encodingTemp) << 1;
			//flip first 1
			(*encodingTemp) = (*encodingTemp) | oneflip;
			//shift left once
			(*encodingTemp) = (*encodingTemp) << 1;
			//flip first 1
			(*encodingTemp) = (*encodingTemp) | oneflip;
		}
		//default N value - 001
		else {
			//shift once
			(*encodingTemp) = (*encodingTemp) << 1;
			//flip the bit 
			(*encodingTemp) = (*encodingTemp) | oneflip;
			//shift twice
			(*encodingTemp) = (*encodingTemp) << 1;
			(*encodingTemp) = (*encodingTemp) << 1;
		}
	}


}

__host__ __device__ void encodeData(char *val, int KSize, dataList *kmerHead, dataList *lmerHead){ 


	/*
		A = 000;
		C = 001;
		G = 010;
		T = 011;
		N = 100;
	
	*/
	//L = our K-mers which are of size K
	//K = our K - 1 mers which are of size K-1	
	struct encodedData *lMer = new encodedData();
	int LSize = KSize;              		//reduce the size sine L = K - 1
	baseToLong(val, LSize, lMer);       //transform the bases into the long long representation
	(*lmerHead).addNewVal(lMer);

	////kmer----------------------------------------------------------------------------
	struct encodedData *kMer = new encodedData();        //create the var that will be the K-Mer
	int KSize1 = LSize - 1;
	baseToLong(val, KSize1, kMer);       //transform the bases into the long long representation
	(*kmerHead).addNewVal(kMer);


	//next step is to put the encoded K-mer into the hash table
	


}




