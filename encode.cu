
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include <inttypes.h>
#include <bitset>
__host__ __device__ void baseToLong(char *val, int KSize, long long unsigned *result){
    /*
        A = 00;
        C = 01;
        G = 10;
        T = 11;
    */
    long long unsigned oneflip = 1;
    // long long unsigned result = 0;

    for (int i = 0; i < KSize; i++) {
        int aciiVal = int(val[i]);
        //A val
        if (aciiVal == 65 || aciiVal == 97) {
            //shift left twice
            (*result) = (*result) << 1;
            (*result) = (*result) << 1;
        }
        //C val
        else if (aciiVal == 67 || aciiVal == 99) {
            //shift left twice
            (*result) = (*result) << 1;
            (*result) = (*result) << 1;
            //flip last bit
            (*result) = (*result) | oneflip;

        }
        //G val
        else if (aciiVal == 71 || aciiVal == 103) {
            //shift left once
            (*result) = (*result) << 1;
            //flip first 1
            (*result) = (*result) | 1;
            //shit left again
            (*result) = (*result) << 1;
        }
        //T val
        else if (aciiVal == 84 || aciiVal == 116) {
            //shift left once
            (*result) = (*result) << 1;
            //flip first 1
            (*result) = (*result) | 1;
            //shift left once
            (*result) = (*result) << 1;
            //flip first 1
            (*result) = (*result) | 1;
        }
        //default bad val
        else{
            printf("Bad Input Flag Triggered \n");
            break;
        }
    }


}

__host__ __device__ void encodData(char *val, int KSize){ 
    long long unsigned kMer = 0;        //create the var that will be the K-Mer
	baseToLong(val, KSize, (&kMer));       //transform the bases into the long long representation

    long long unsigned lMer = 0;         //create the var that will be the L-Mer
    int LSize = KSize - 1;              //reduce the size sine L = K - 1
    baseToLong(val, LSize, (&lMer));       //transform the bases into the long long representation
    
    //next step is to put the encoded K-mer into the hash table
}